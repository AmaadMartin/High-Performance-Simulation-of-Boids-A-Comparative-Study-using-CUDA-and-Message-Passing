#include "hip/hip_runtime.h"
#include "Boid.h"
#include "CudaFlock.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


// =============================================== //
// ======== Flock Functions from CudaFlock.h ========= //
// =============================================== //

int CudaFlock::getSize()
{
    return flock.size();
}

Boid CudaFlock::getBoid(int i)
{
    return flock[i];
}

vector<Boid> CudaFlock::getFlock()
{
    return flock;
}

void CudaFlock::addBoid(const Boid& b)
{
    flock.push_back(std::move(b));
}

__global__ void cudaFlocking(Boid* flock, int flockSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < flockSize) {
        flock[idx].run(flock, flockSize);
    }
}

void CudaFlock::flocking() 
{
    int size = flock.size();

    Boid* deviceFlock;
    int numBytes = size * sizeof(Boid);
    
    // Allocate device memory for flock
    hipMalloc((void**)&deviceFlock, numBytes);
    
    // Copy flock from host to device
    hipMemcpy(deviceFlock, flock.data(), numBytes, hipMemcpyHostToDevice);
    
    // Run cudaFlocking kernel
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    cudaFlocking<<<numBlocks, blockSize>>>(deviceFlock, size);
    
    // Copy flock from device to host
    hipMemcpy(flock.data(), deviceFlock, numBytes, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(deviceFlock);
}
