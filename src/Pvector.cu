#include "hip/hip_runtime.h"
#include <math.h>
#include "Pvector.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// =================================================== //
// ======== Pvector Functions from Pvector.h ========= //
// =================================================== //

#define PI 3.141592635

// Sets values of x and y for Pvector
__device__ void Pvector::set(float i, float o)
{
    x = i;
    y = o;
}

__device__ void Pvector::addVector(const Pvector& v)
{
    x += v.x;
    y += v.y;
}

// Adds to a Pvector by a constant number
__device__ void Pvector::addScalar(float s)
{
    x += s;
    y += s;
}

// Subtracts 2 vectors
__device__ void Pvector::subVector(const Pvector& v)
{
    x -= v.x;
    y -= v.y;
}

// Subtracts two vectors and returns the difference as a vector
__device__ Pvector Pvector::subTwoVector(const Pvector& v, const Pvector& v2)
{
    Pvector tmp(v.x - v2.x, v.y - v2.y);
    return std::move(tmp);
}

// Adds to a Pvector by a constant number
__device__ void Pvector::subScalar(float s)
{
    x -= s;
    y -= s;
}

// Multiplies 2 vectors
__device__ void Pvector::mulVector(const Pvector& v)
{
    x *= v.x;
    y *= v.y;
}

// Adds to a Pvector by a constant number
__device__ void Pvector::mulScalar(float s)
{
    x *= s;
    y *= s;
}

// Divides 2 vectors
__device__ void Pvector::divVector(const Pvector& v)
{
    x /= v.x;
    y /= v.y;
}

// Adds to a Pvector by a constant number
__device__ void Pvector::divScalar(float s)
{
    x /= s;
    y /= s;
}

__device__ void Pvector::limit(double max)
{
    double size = magnitude();

    if (size > max) {
        set(x / size, y / size);
    }
}

// Calculates the distance between the first Pvector and second Pvector
__device__ float Pvector::distance(const Pvector& v)
{
    float dx = x - v.x;
    float dy = y - v.y;
    float dist = sqrt(dx*dx + dy*dy);
    return dist;
}

// Calculates the dot product of a vector
__device__ float Pvector::dotProduct(const Pvector& v)
{
    float dot = x * v.x + y * v.y;
    return dot;
}

// Calculates magnitude of referenced object
__device__ float Pvector::magnitude()
{
    return sqrt(x*x + y*y);
}

__device__ void Pvector::setMagnitude(float x)
{
    normalize();
    mulScalar(x);
}

// Calculate the angle between Pvector 1 and Pvector 2
__device__ float Pvector::angleBetween(const Pvector& v)
{
    if (x == 0 && y == 0) return 0.0f;
    if (v.x == 0 && v.y == 0) return 0.0f;

    double dot = x * v.x + y * v.y;
    double v1mag = sqrt(x * x + y * y);
    double v2mag = sqrt(v.x * v.x + v.y * v.y);
    double amt = dot / (v1mag * v2mag); //Based of definition of dot product
    //dot product / product of magnitudes gives amt
    if (amt <= -1) {
        return PI;
    } else if (amt >= 1) {
        return 0;
    }
    float tmp = acos(amt);
    return tmp;
}

// normalize divides x and y by magnitude if it has a magnitude.
__device__ void Pvector::normalize()
{
    float m = magnitude();

    if (m > 0) {
        set(x / m, y / m);
    } else {
        set(x, y);
    }
}

// Creates and returns a copy of the Pvector used as a parameter
__device__ Pvector Pvector::copy(const Pvector& v)
{
    Pvector copy(v.x, v.y);
    return copy;
}
